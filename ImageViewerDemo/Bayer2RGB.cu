#include "hip/hip_runtime.h"

#include<iostream>
#include <hip/hip_runtime.h> 
#include <>

/*�˺������豸���к�����*/
__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x; //��ȡ�߳�id
	if (i < numElements)
	{
		C[i] = A[i] * B[i] + 10;
	}
}
//�����ں˺��������豸���߳�������
__global__ void VecAdd(float* a, float* b, float* c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void addOne(int* data, int numdata)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x; //��ȡ�߳�id
	if (i < numdata)
	{
		data[i] = data[i] + 1;
	}
}


void test()
{
	float A[5] = { 1.0,2.2,3.3,4.4,5.5 };
	float B[5] = { 1.0,2.2,3.3,4.4,5.5 };
	float C[5] = { 1.0,2.2,3.3,4.4,5.5 };
	VecAdd << <1, 5 >> > (A, B, C);
	std::cout << A[0] << std::endl;
}

/*��������*/
void test1(int num)
{
	/*�������������ڴ� h_A, h_B, h_C*/
	int numElements = num;
	size_t size = numElements * sizeof(float);
	float* h_A = (float*)malloc(size);
	float* h_B = (float*)malloc(size);
	float* h_C = (float*)malloc(size);
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	/*�����豸�ڴ� d_A��d_B��d_C */
	float* d_A = NULL;
	hipMalloc((void**)&d_A, size);
	float* d_B = NULL;
	hipMalloc((void**)&d_B, size);
	float* d_C = NULL;
	hipMalloc((void**)&d_C, size);

	/*�������ڴ����ݸ��Ƶ��豸�ڴ� h_A--d_A��h_B--d_B */
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	/*�����豸���߳����������ú˺���*/
	int threadsPerBlock = 256; //ÿ���߳̿���߳�����
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock; //�߳̿������
	vectorAdd << < blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
	hipGetLastError();

	/*���豸�ڴ����ݸ��Ƶ������ڴ� d_C--h_C */
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	/* �ͷ��豸�ڴ� d_A d_B d_C */
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	/* �����֤ */
	std::cout << "A[0]: " << (float)h_A[0] << " B[0]: " << (float)h_B[0] << " ���C[0] = A[i] + B[i] + 10: " << (float)h_C[0] << std::endl;
	std::cout << "A[1]: " << (float)h_A[1] << " B[1]: " << (float)h_B[1] << " ���C[1] = A[i] + B[i] + 10: " << (float)h_C[1] << std::endl;
	std::cout << "A[2]: " << (float)h_A[2] << " B[2]: " << (float)h_B[2] << " ���C[2] = A[i] + B[i] + 10: " << (float)h_C[2] << std::endl;

	/* �ͷ������ڴ� h_A h_B h_C */
	free(h_A);
	free(h_B);
	free(h_C);
}


void addList(int* _data_ptr, int size)
{
	size_t data_size = size * sizeof(int);
	int* d_A = NULL;
	hipMalloc((void**)&d_A, data_size);

	hipMemcpy(d_A, _data_ptr, data_size, hipMemcpyHostToDevice);
	/*�����豸���߳����������ú˺���*/
	int threadsPerBlock = 256; //ÿ���߳̿���߳�����
	int blocksPerGrid = (data_size + threadsPerBlock - 1) / threadsPerBlock; //�߳̿������
	addOne << < blocksPerGrid, threadsPerBlock >> > (d_A, data_size);

	/*���豸�ڴ����ݸ��Ƶ������ڴ� d_C--h_C */
	hipMemcpy(_data_ptr, d_A, data_size, hipMemcpyDeviceToHost);

	hipFree(d_A);
}